/**
 * @file test_matrix_relu.cu
 * @brief Unit tests for the Matrix::relu method.
 */

#include <gtest/gtest.h>
#include "../src/linear_algebra/matrix.h"
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @class MatrixReluTest
 * @brief Test fixture for the Matrix::relu method tests.
 */
class MatrixReluTest : public ::testing::Test {
protected:
    void SetUp() override {}
    void TearDown() override {}
};

/**
 * @test
 * @brief Verify that Matrix::relu correctly applies the ReLU function.
 *
 * This test creates a matrix with positive and negative values,
 * applies the ReLU function, and confirms that all negative values
 * are set to zero while positive values remain unchanged.
 */
TEST_F(MatrixReluTest, ApplyReluCorrectly) {
    // Create a 3x3 matrix with known values
    Matrix m(3, 3);
    double h_data[9] = {-1.0, 0.0, 1.0, -2.0, 3.0, -4.0, 5.0, -6.0, 7.0};
    hipMemcpy(m.get_data(), h_data, 9 * sizeof(double), hipMemcpyHostToDevice);

    // Apply ReLU to the matrix
    Matrix result = m.relu();

    // Print the original and result matrices
    std::cout << "Original matrix:" << std::endl;
    m.print(2);
    std::cout << "After ReLU:" << std::endl;
    result.print(2);

    // Allocate host memory to verify the results
    double* h_result = new double[9];
    hipMemcpy(h_result, result.get_data(), 9 * sizeof(double), hipMemcpyDeviceToHost);

    // Define the expected result after ReLU
    double expected[9] = {0.0, 0.0, 1.0, 0.0, 3.0, 0.0, 5.0, 0.0, 7.0};

    // Check that all elements are correctly processed by ReLU
    for (int i = 0; i < 9; ++i) {
        EXPECT_DOUBLE_EQ(h_result[i], expected[i]);
    }

    // Free the allocated host memory
    delete[] h_result;
}
