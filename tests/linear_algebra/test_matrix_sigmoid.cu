#include "hip/hip_runtime.h"
/**
 * @file test_matrix_sigmoid.cu
 * @brief Unit tests for the Matrix::sigmoid method.
 */
#include <gtest/gtest.h>
#include "../../src/linear_algebra/matrix.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

/**
 * @class MatrixSigmoidTest
 * @brief Test fixture for the Matrix::sigmoid method tests.
 */
class MatrixSigmoidTest : public ::testing::Test {
protected:
    void SetUp() override {}
    void TearDown() override {}
};

/**
 * @test
 * @brief Verify that Matrix::sigmoid correctly applies the sigmoid function.
 *
 * This test creates a matrix with various values,
 * applies the sigmoid function, and confirms that the output
 * is within the expected range and matches calculated values.
 */
TEST_F(MatrixSigmoidTest, ApplySigmoidCorrectly) {
    // Create a 3x3 matrix with known values
    Matrix m(3, 3);
    double h_data[9] = {-2.0, -1.0, 0.0, 0.5, 1.0, 2.0, 3.0, 4.0, 5.0};
    hipMemcpy(m.get_data(), h_data, 9 * sizeof(double), hipMemcpyHostToDevice);

    // Apply sigmoid to the matrix
    Matrix result = m.sigmoid();

    // Print the original and result matrices
    std::cout << "Original matrix:" << std::endl;
    m.print(2);
    std::cout << "After sigmoid:" << std::endl;
    result.print(4);

    // Allocate host memory to verify the results
    double* h_result = new double[9];
    hipMemcpy(h_result, result.get_data(), 9 * sizeof(double), hipMemcpyDeviceToHost);

    // Check that all elements are correctly processed by sigmoid
    for (int i = 0; i < 9; ++i) {
        // Calculate expected sigmoid value
        double expected = 1.0 / (1.0 + std::exp(-h_data[i]));
        
        // Check if the result is within a small epsilon of the expected value
        EXPECT_NEAR(h_result[i], expected, 1e-6);
        
        // Check if the result is within the valid range for sigmoid (0 to 1)
        EXPECT_GE(h_result[i], 0.0);
        EXPECT_LE(h_result[i], 1.0);
    }

    // Free the allocated host memory
    delete[] h_result;
}
