#include "hip/hip_runtime.h"
/**
 * @file test_matrix_divide_scalar.cu
 * @brief Unit tests for the Matrix::divide_scalar method.
 */

#include <gtest/gtest.h>
#include "../src/linear_algebra/matrix.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>

/**
 * @class MatrixDivideScalarTest
 * @brief Test fixture for the Matrix::divide_scalar method tests.
 */
class MatrixDivideScalarTest : public ::testing::Test {
protected:
    void SetUp() override {}
    void TearDown() override {}

    // Helper function to check if two doubles are approximately equal
    bool isApproximatelyEqual(double a, double b, double epsilon = 1e-6) {
        if (std::isinf(a) && std::isinf(b)) {
            return (a > 0) == (b > 0);
        }
        if (std::abs(a) > DBL_MAX / 2 || std::abs(b) > DBL_MAX / 2) {
            // For very large numbers, use a relative error
            return std::abs(a - b) / std::max(std::abs(a), std::abs(b)) < epsilon;
        }
        return std::abs(a - b) < epsilon;
    }
};

/**
 * @test
 * @brief Verify that Matrix::divide_scalar correctly divides all elements by a scalar.
 */
TEST_F(MatrixDivideScalarTest, DivideMatrixByScalarCorrectly) {
    // Create test matrix
    Matrix m(2, 3);

    // Prepare test data
    double h_m[6] = {2.0, 4.0, 6.0, 8.0, 10.0, 12.0};

    // Copy test data to GPU
    hipMemcpy(m.get_data(), h_m, 6 * sizeof(double), hipMemcpyHostToDevice);

    // Print original matrix
    std::cout << "Original matrix:" << std::endl;
    m.print(2);

    // Perform division by scalar
    double scalar = 2.0;
    m.divide_scalar(scalar);

    // Print result
    std::cout << "Matrix after dividing by " << scalar << ":" << std::endl;
    m.print(2);

    // Copy result back to CPU for verification
    double* h_result = new double[6];
    hipMemcpy(h_result, m.get_data(), 6 * sizeof(double), hipMemcpyDeviceToHost);

    // Define expected result
    double expected[6] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};

    // Verify result
    for (int i = 0; i < 6; ++i) {
        EXPECT_DOUBLE_EQ(h_result[i], expected[i]);
    }

    // Clean up
    delete[] h_result;
}

/**
 * @test
 * @brief Verify that Matrix::divide_scalar handles division by a very small number correctly.
 */
TEST_F(MatrixDivideScalarTest, DivideByVerySmallNumber) {
    // Create test matrix
    Matrix m(2, 2);
    
    // Prepare test data
    double h_m[4] = {1.0, -1.0, 0.0, 2.0};
    hipMemcpy(m.get_data(), h_m, 4 * sizeof(double), hipMemcpyHostToDevice);

    // Print original matrix
    std::cout << "Original matrix:" << std::endl;
    m.print(2);

    // Perform division by a very small number
    double scalar = DBL_EPSILON / 2.0;  // Very small number
    std::cout << "Dividing by scalar: " << scalar << std::endl;
    m.divide_scalar(scalar);

    // Print result
    std::cout << "Matrix after dividing by very small number:" << std::endl;
    m.print(2);

    // Copy result back to CPU for verification
    double* h_result = new double[4];
    hipMemcpy(h_result, m.get_data(), 4 * sizeof(double), hipMemcpyDeviceToHost);

    // Verify results
    EXPECT_DOUBLE_EQ(h_result[0], DBL_MAX);
    EXPECT_DOUBLE_EQ(h_result[1], -DBL_MAX);
    EXPECT_DOUBLE_EQ(h_result[2], 0.0);
    EXPECT_DOUBLE_EQ(h_result[3], DBL_MAX);

    // Clean up
    delete[] h_result;
}

/**
 * @test
 * @brief Verify that Matrix::divide_scalar handles division of very large numbers correctly.
 */
TEST_F(MatrixDivideScalarTest, DivideLargeNumbers) {
    // Create test matrix
    Matrix m(2, 2);
    
    // Prepare test data with very large numbers
    double h_m[4] = {DBL_MAX, -DBL_MAX, DBL_MAX / 2, -DBL_MAX / 2};
    hipMemcpy(m.get_data(), h_m, 4 * sizeof(double), hipMemcpyHostToDevice);

    // Print original matrix
    std::cout << "Original matrix with large numbers:" << std::endl;
    m.print(4);

    // Perform division
    double scalar = 2.0;
    std::cout << "Dividing by scalar: " << scalar << std::endl;
    m.divide_scalar(scalar);

    // Print result
    std::cout << "Matrix after dividing large numbers:" << std::endl;
    m.print(4);

    // Copy result back to CPU for verification
    double* h_result = new double[4];
    hipMemcpy(h_result, m.get_data(), 4 * sizeof(double), hipMemcpyDeviceToHost);

    // Verify results using approximate equality
    EXPECT_TRUE(isApproximatelyEqual(h_result[0], DBL_MAX, 1e-6));
    EXPECT_TRUE(isApproximatelyEqual(h_result[1], -DBL_MAX, 1e-6));
    EXPECT_TRUE(isApproximatelyEqual(h_result[2], DBL_MAX / 4, 1e-6));
    EXPECT_TRUE(isApproximatelyEqual(h_result[3], -DBL_MAX / 4, 1e-6));

    // Clean up
    delete[] h_result;
}

/**
 * @test
 * @brief Verify that Matrix::divide_scalar throws an exception when dividing by exactly zero.
 */
TEST_F(MatrixDivideScalarTest, ThrowsExceptionWhenDividingByExactlyZero) {
    // Create test matrix
    Matrix m(2, 2);
    
    // Attempt to divide by zero and expect an exception
    EXPECT_THROW(m.divide_scalar(0.0), std::invalid_argument);
    
    std::cout << "Successfully caught exception when dividing by zero." << std::endl;
}
