#include "hip/hip_runtime.h"
/**
 * @file matrix_sigmoid.cu
 * @brief Implementation of the sigmoid activation function for matrices.
 */

#include "matrix.h"
#include <hip/hip_runtime.h>
#include <math.h>

/**
 * @brief CUDA kernel for applying the sigmoid activation function element-wise.
 * @param input Pointer to the input matrix data.
 * @param output Pointer to the output matrix data.
 * @param size Total number of elements in the matrix.
 */
__global__ void sigmoidKernel(const double* input, double* output, int size) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the thread is within the matrix bounds
    if (idx < size) {
        // Apply sigmoid: 1 / (1 + exp(-x))
        output[idx] = 1.0 / (1.0 + exp(-input[idx]));
    }
}

/**
 * @brief Applies the sigmoid activation function to the matrix.
 * @return A new Matrix object with sigmoid applied.
 */
Matrix Matrix::sigmoid() const {
    // Create a new matrix with the same dimensions
    Matrix result(rows, cols);
    
    // Calculate the total number of elements
    int size = rows * cols;
    
    // Define the number of threads per block
    int threadsPerBlock = 256;
    
    // Calculate the number of blocks needed
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    // Launch the CUDA kernel
    sigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, result.d_data, size);
    
    // Synchronize to ensure the kernel execution is complete
    hipDeviceSynchronize();
    
    return result;
}
