/**
 * @file vector_initialize.cu
 * @brief Implementation of the Vector::initialize method.
 */
#include "vector.h"
#include <hip/hip_runtime.h>

void Vector::initialize() {
    // Use hipMemset to set all elements of d_data to 0
    hipMemset(d_data, 0, rows * sizeof(double));
}
