/**
 * @file matrix_copy.cu
 * @brief Implementation of copy and move operations for the Matrix class.
 */

#include "matrix.h"
#include <hip/hip_runtime.h>

Matrix::Matrix(const Matrix& other) : rows(other.rows), cols(other.cols) {
    // Allocate new memory on the device
    hipMalloc(&d_data, rows * cols * sizeof(double));
    // Copy data from the other matrix to this one
    hipMemcpy(d_data, other.d_data, rows * cols * sizeof(double), hipMemcpyDeviceToDevice);
}

Matrix& Matrix::operator=(const Matrix& other) {
    if (this != &other) {  // Protect against self-assignment
        // Free existing memory
        hipFree(d_data);
        
        // Copy dimensions
        rows = other.rows;
        cols = other.cols;
        
        // Allocate new memory
        hipMalloc(&d_data, rows * cols * sizeof(double));
        // Copy data from the other matrix
        hipMemcpy(d_data, other.d_data, rows * cols * sizeof(double), hipMemcpyDeviceToDevice);
    }
    return *this;
}

Matrix::Matrix(Matrix&& other) noexcept
    : rows(other.rows), cols(other.cols), d_data(other.d_data) {
    // Transfer ownership and reset the source object
    other.d_data = nullptr;
    other.rows = 0;
    other.cols = 0;
}

Matrix& Matrix::operator=(Matrix&& other) noexcept {
    if (this != &other) {  // Protect against self-assignment
        // Free existing memory
        hipFree(d_data);
        
        // Transfer ownership
        rows = other.rows;
        cols = other.cols;
        d_data = other.d_data;
        
        // Reset the source object
        other.d_data = nullptr;
        other.rows = 0;
        other.cols = 0;
    }
    return *this;
}

Matrix Matrix::copy() const {
    // Use the copy constructor to create a deep copy
    return *this;
}
