/**
 * @file matrix_initialize.cu
 * @brief Implementation of the Matrix::initialize method.
 */
#include "matrix.h"
#include <hip/hip_runtime.h>

void Matrix::initialize() {
    // Use hipMemset to set all elements of d_data to 0
    hipMemset(d_data, 0, rows * cols * sizeof(double));
}
